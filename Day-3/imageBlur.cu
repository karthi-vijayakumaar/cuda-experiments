#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <vector>
#include <hip/hip_runtime.h>

void readVectorFromFile(const char* filename, std::vector<int>& vec) {
    std::ifstream file(filename);
    int value;
    while (file >> value) {
        vec.push_back(value);
    }
}

__global__ void blur(int *a, int *b, int height, int width) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
        if (row < height && col < width){
        int sum_of_surround = 0;
        int num_of_surround = 0;
        for(i=-1;i<=1;i++){
            for(j=-1;j<=1;j++){
            if (row+i>=0 && row+i < height && col+j >=0 && col+j < width) {
                    int offset = (row+i) * width + (col+j);
                    sum_of_surround += a[offset];
                    num_of_surround++;

                } 
            }
        }
        b[row * width + col] = sum_of_surround / num_of_surround;
    }
    
}

int main() {
    const char *filename1 = "image.txt";

    std::vector<int> vector1, result;

    readVectorFromFile(filename1, vector1);

    int n = vector1.size();
    int height=210, width=236;

    std::cout << "size of the vector is " << n << std::endl;
    std::cout << "size of the image is " << height << "x" << width << std::endl;

    int *d_1, *d_result;

    hipMalloc((void**)&d_1, n * sizeof(int));
    hipMalloc((void**)&d_result, n * sizeof(int));

    hipMemcpy(d_1, vector1.data(), n * sizeof(int), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((width + threadsPerBlock.x - 1) / threadsPerBlock.x, (height + threadsPerBlock.y - 1) / threadsPerBlock.y);

    blur<<<blocksPerGrid, threadsPerBlock>>>(d_1, d_result, height, width);
    result.resize(n);
    hipMemcpy(result.data(), d_result, n * sizeof(int), hipMemcpyDeviceToHost);

    // Store the result in a space separated txt file
    std::ofstream file("result.txt");
    for (int i = 0; i < n; i++) {
        file << result[i] << " ";
    }
    file.close();

    std::cout << "Result of color to bw conversion:\n";
    for (int i = 0; i < n; i++) {
        std::cout <<result[i] << " ";
    }

    // Free device memory
    hipFree(d_1);
    hipFree(d_result);
}